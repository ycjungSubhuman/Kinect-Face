#include "hip/hip_runtime.h"
#include "face/cu_model_kernel.h"

#include <iostream>
#include <cmath>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <vector>
#include <map>

/* Includes, cuda */
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "util/cudautil.h"
#include "util/cu_quaternion.h"
#include "align/cu_loss.h"
#include "util/transform.h"

#define BLOCKSIZE 128

__global__
void _calculateVertexPosition(float *position_d, const C_Params params, const C_PcaDeformModel deformModel) {
    int start_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // total number of threads in the grid

    const int colDim = deformModel.dim;

    // grid-striding loop
    for (int i = start_index; i < deformModel.dim; i += stride) {

        position_d[i] = 0;
        for (int j = 0; j < deformModel.shapeRank; j++) {
            position_d[i] += params.fa1Params_d[j] * deformModel.shapeDeformBasis_d[i + colDim * j];
        }

        for (int j = 0; j < deformModel.expressionRank; j++) {
            position_d[i] += params.fa2Params_d[j] * deformModel.expressionDeformBasis_d[i + colDim * j];
        }

        position_d[i] +=
                deformModel.meanShapeDeformation_d[i]
                + deformModel.meanExpressionDeformation_d[i]
                + deformModel.ref_d[i];
    }
}

void calculateVertexPosition(float *position_d, const C_Params params, const C_PcaDeformModel deformModel) {
    int idim = deformModel.dim;
    dim3 dimBlock(BLOCKSIZE);
    dim3 dimGrid((idim + BLOCKSIZE - 1) / BLOCKSIZE);

    _calculateVertexPosition << < dimGrid, dimBlock >> > (position_d, params, deformModel);
    CHECK_ERROR_MSG("Kernel Error");
}

__global__
void _homogeneousPositions(float *h_position_d, const float *position_d, int nPoints) {

    int start_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // total number of threads in the grid

    // grid-striding loop
    for (int index = start_index; index < nPoints; index += stride) {
        // homogeneous coordinates (x,y,z,1);
        float pos[4] = {position_d[3 * index], position_d[3 * index + 1], position_d[3 * index + 2], 1};
        memcpy(&h_position_d[4 * index], &pos[0], 4 * sizeof(float));
    }
}

__global__
void _hnormalizedPositions(float *position_d, const float *h_position_d, int nPoints) {

    int start_index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // total number of threads in the grid

    // grid-striding loop
    for (int index = start_index; index < nPoints; index += stride) {

        // homogeneous coordinates (x,y,z,1);
        float hnorm = h_position_d[4 * index + 3];
        position_d[3 * index] = h_position_d[4 * index] / hnorm;
        position_d[3 * index + 1] = h_position_d[4 * index + 1] / hnorm;
        position_d[3 * index + 2] = h_position_d[4 * index + 2] / hnorm;
    }
}

void cudaMatMul(float *matC, hipblasHandle_t cnpHandle,
                const float *matA, int aRows, int aCols,
                const float *matB, int bRows, int bCols) {

    // Don't know what this is (scalar?) but examples use this
    hipblasStatus_t status;
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;

    /* Perform operation using cublas, inputs/outputs are col-major.
     * vector and array were originally Eigen which defaults to Col-major
     * m is rows for A and C
     * n is cols for B and C
     * k is cols for A and rows for B*/
    // Matrix Mult C = α op ( A ) op ( B ) + β C
    status =
            hipblasSgemm(cnpHandle,
                        HIPBLAS_OP_N, HIPBLAS_OP_N, // Matrix op(A) and op(B): No-op, Transpose, Conjugate
                        aRows, bCols, aCols, //(m,n,k)
                        alpha,
                        matA, aRows/*leading dim, ROWS?*/, //(4x4) or (mxk)
                        matB, bRows/*leading dim*/, //(4xN) or (kxn)
                        beta,
                        matC, bRows/*leading dim*/); //(4xN) or (mxk)

    if (status != HIPBLAS_STATUS_SUCCESS) {
      printf("SGEMM a b : (%d,%d), (%d,%d)\n", aRows, aCols, bRows, bCols);
      printf("SGEMM status: %d\n", status);
        throw std::runtime_error("MatMul Failed\n");
    }
}

void applyRigidAlignment(float *align_pos_d, hipblasHandle_t cnpHandle,
                         const float *position_d, const float *transMat, int N) {
    int size_homo = 4 * N;
    dim3 grid = ((N + BLOCKSIZE - 1) / BLOCKSIZE);
    dim3 block = BLOCKSIZE;

    float *matB, *matC;

    hipMalloc((void **) &matB, size_homo * sizeof(float));
    hipMalloc((void **) &matC, size_homo * sizeof(float));


    // Create homogenous matrix (x,y,z,1)
    _homogeneousPositions << < grid, block >> > (matB, position_d, N);
    CHECK_ERROR_MSG("Kernel Error");

    /* Perform operation using cublas, inputs/outputs are col-major.
     * vector and array were originally Eigen which defaults to Col-major
     * m is rows for A and C
     * n is cols for B and C
     * k is cols for A and rows for B*/
    // Matrix Mult C = α op ( A ) op ( B ) + β C
    cudaMatMul(matC, cnpHandle, transMat, 4, 4, matB, 4, N);

    // hnormalized point (x,y,z)
    _hnormalizedPositions << < grid, block >> > (align_pos_d, matC, N);
    CHECK_ERROR_MSG("Kernel Error");

    hipFree(matB);
    hipFree(matC);
}

__global__
static void calculateLandmarkIndices(int *mesh_inds, int *scan_inds, C_PcaDeformModel model, C_ScanPointCloud scan) {
    const int start = blockDim.x * blockIdx.x + threadIdx.x;
    const int size = scan.numLmks;
    const int step = blockDim.x * gridDim.x;

    for(int ind=start; ind<size; ind+=step) {
        mesh_inds[ind] = model.lmks_d[scan.modelLandmarkSelection_d[ind]];
        scan_inds[ind] = scan.modelLandmarkSelection_d[ind];
    }
}

/**
 * Project xyz coord into uv space
 * @param uv
 * @param xyz
 * @param fx
 * @param fy
 * @param cx
 * @param cy
 */
__device__
void convertXyzToUv(int *uv, const float* xyz, float fx, float fy, float cx, float cy) {
    uv[0] = static_cast<int>(std::round(xyz[0] * fx / xyz[2] + cx));
    uv[1] = static_cast<int>(std::round(xyz[1] * fy / xyz[2] + cy));
}

__global__
void _find_mesh_to_scan_corr(int *meshCorr_d, int *scanCorr_d, float *distance_d, int *numCorr,
                             const float *position_d, int num_points, C_ScanPointCloud scan, float radius, int maxPoints) {
    const int start = blockIdx.x * blockDim.x + threadIdx.x;
    const int size = num_points;
    const int step = blockDim.x * gridDim.x;
    // Initialize numCorr to 0, will use atomicAdd to increment counter
    if(threadIdx.x == 0) numCorr[0] = 0;
    __syncthreads();

    for(int i=start; i<size; i+=step) {
        // Project Point into UV space for mapping finding closest xy on scan
        int uv[2];

        convertXyzToUv(&uv[0], &position_d[i*3], scan.fx, scan.fy, scan.cx, scan.cy);

        // Calculate 1-D Index of scan point from UV coord
        int scanIndx = uv[1] * scan.width * 3 + uv[0] * 3 + 0;
        int scanIndy = uv[1] * scan.width * 3 + uv[0] * 3 + 1;
        int scanIndz = uv[1] * scan.width * 3 + uv[0] * 3 + 2;

        // Check if Model coord outside of UV space, could happen if model is aligned to face near image boarder
        if(scanIndx < scan.numPoints*3 && scanIndy < scan.numPoints*3 && scanIndz < scan.numPoints*3 &&
           scanIndx >=0 && scanIndy >=0 && scanIndz >=0) {
            // Check for NaN Points
            bool isNaN = std::isfinite(scan.scanPoints_d[scanIndx]) == 0
                         || std::isfinite(scan.scanPoints_d[scanIndy]) == 0
                         || std::isfinite(scan.scanPoints_d[scanIndz]) == 0;

            // Add correspondance if within search radius, if radius is 0, include all points
            if (!isNaN) {
                // Check z distance for within radius tolerance (Use xyz EuclidDist instead?)
                float dist = std::fabs(position_d[i * 3 + 2] - scan.scanPoints_d[scanIndx + 2]);
//                printf("Correspondance %.4f\n", dist);
                if (radius <= 0 || dist <= radius) {
                    int idx = atomicAdd(&numCorr[0], 1);
                    if (maxPoints <= 0 || idx < maxPoints) {
//                        printf("Correspondance s:%d -> m:%d, d:%.4f\n", scanIndx / 3, i, dist);
                        meshCorr_d[idx] = i;
                        scanCorr_d[idx] = scanIndx / 3;
                        distance_d[idx] = dist;
                    }
                }
            }
        }
    }
}

void reduce_closest_corr(int *meshCorr_d, int *scanCorr_d, float *distance_d, int *numCorr_d, int maxPoints) {
    int numCorr;
    CUDA_CHECK(hipMemcpy(&numCorr, numCorr_d, sizeof(int), hipMemcpyDeviceToHost));

    if (numCorr > maxPoints){
        numCorr = maxPoints;
    }

    int *meshCorr_h = new int[numCorr];
    int *scanCorr_h = new int[numCorr];
    float *distance_h = new float[numCorr];

    CUDA_CHECK(hipMemcpy(meshCorr_h, meshCorr_d, numCorr * sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(scanCorr_h, scanCorr_d, numCorr * sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(distance_h, distance_d, numCorr * sizeof(float), hipMemcpyDeviceToHost));

    std::map<int,int> scantomesh;
    std::map<int,float> scandist;

    for (int idx = 0; idx < numCorr; idx++){
        auto meshIdx =  meshCorr_h[idx];
        auto scanIdx =  scanCorr_h[idx];
        auto dist =  distance_h[idx];

        if(scantomesh.find(scanIdx) != scantomesh.end()) {
            if (dist < scandist[scanIdx]){
                scantomesh[scanIdx] = meshIdx;
                scandist[scanIdx] = dist;
            }
        } else {
            scantomesh.insert(std::make_pair(scanIdx, meshIdx));
            scandist.insert(std::make_pair(scanIdx, dist));
        }
    }

    std::vector<int> finScan, finMesh;
    std::vector<float> findist;
    for(std::map<int,int>::iterator it = scantomesh.begin(); it != scantomesh.end(); ++it) {
        finScan.push_back(it->first);
        finMesh.push_back(it->second);
        findist.push_back(scandist[it->first]);
    }
    auto size = finMesh.size();
    CUDA_CHECK(hipMemcpy(meshCorr_d, finMesh.data(), size* sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(scanCorr_d, finScan.data(), size* sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(distance_d, findist.data(), size* sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(numCorr_d, &size, sizeof(int), hipMemcpyHostToDevice));
}

void find_mesh_to_scan_corr(int *meshCorr_d, int *scanCorr_d, float *distance_d, int *numCorr,
                            const float *position_d, int num_points, C_ScanPointCloud scan, float radius, int maxPoints) {
    int idim = num_points/3;
    dim3 dimBlock(BLOCKSIZE);
    dim3 dimGrid((idim + BLOCKSIZE - 1) / BLOCKSIZE);
    CUDA_ZERO(&numCorr, static_cast<size_t >(1));

    _find_mesh_to_scan_corr << < dimGrid, dimBlock >> > (meshCorr_d, scanCorr_d, distance_d, numCorr,
            position_d, num_points, scan, radius, maxPoints);
    reduce_closest_corr(meshCorr_d, scanCorr_d, distance_d, numCorr, maxPoints);
    CHECK_ERROR_MSG("Kernel Error");
}

void calculateAlignedPositions(float *result_pos_d, float *align_pos_d, float *position_d,
                               const C_Params params, const C_PcaDeformModel deformModel, const C_ScanPointCloud scanPointCloud,
                               hipblasHandle_t cnpHandle){
    // Calculate position_d
    calculateVertexPosition(position_d, params, deformModel);

    // Rigid alignment
    applyRigidAlignment(align_pos_d, cnpHandle, position_d, scanPointCloud.rigidTransform_d, deformModel.dim / 3);
    float r[9];
    float trans[16];
    float *trans_d;
    CUDA_CHECK(hipMalloc((void **) &trans_d, 16*sizeof(float)));

    calc_r_from_u(r, params.fuParams_h);
    create_trans_from_tu(trans, params.ftParams_h, r);
    CUDA_CHECK(hipMemcpy(trans_d, trans, 16* sizeof(float), hipMemcpyHostToDevice));
    applyRigidAlignment(result_pos_d, cnpHandle, align_pos_d, trans_d, deformModel.dim / 3);
}

void calculatePointPairLoss(float *residual, float *fa1Jacobian, float *fa2Jacobian, float *ftJacobian, float *fuJacobian,
                            PointPair point_pair, C_Params params, C_PcaDeformModel deformModel,
                            C_Residuals c_residuals, C_Jacobians c_jacobians,
                            const float weight, const bool isJacobianRequired) {

    if (point_pair.point_count > 0) {
        calc_residual_point_pair(c_residuals.residual_d, point_pair, weight);

    }

    /*
     * Copy computed residual to Host
     */
    CUDA_CHECK(hipMemcpy(residual, c_residuals.residual_d, c_residuals.numResuduals*sizeof(float), hipMemcpyDeviceToHost));

    if (isJacobianRequired) {
        // Compute Jacobians for each parameter
        if (point_pair.point_count > 0) {
            calc_derivatives_point_pair(c_jacobians.ftJacobian_d, c_jacobians.fuJacobian_d,
                                        c_jacobians.fa1Jacobian_d, c_jacobians.fa2Jacobian_d,
                                        params.fuParams_d, deformModel, point_pair, weight);
        }

        /*
         * Copy computed jacobian to Host
         */
        CUDA_CHECK(hipMemcpy(fa1Jacobian, c_jacobians.fa1Jacobian_d, c_jacobians.numa1j * sizeof(float), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(fa2Jacobian, c_jacobians.fa2Jacobian_d, c_jacobians.numa2j * sizeof(float), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(ftJacobian, c_jacobians.ftJacobian_d, c_jacobians.numtj * sizeof(float), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(fuJacobian, c_jacobians.fuJacobian_d, c_jacobians.numuj * sizeof(float), hipMemcpyDeviceToHost));
    }

}

void calculateLandmarkLoss(float *residual, float *fa1Jacobian, float *fa2Jacobian, float *ftJacobian, float *fuJacobian,
                           float *position_d, hipblasHandle_t cnpHandle, C_Params params, C_PcaDeformModel deformModel,
                           C_ScanPointCloud scanPointCloud, C_Residuals c_residuals, C_Jacobians c_jacobians,
                           const float weight, const bool isJacobianRequired) {

    float *align_pos_d, *result_pos_d;


    // Allocate memory for Rigid aligned positions
    CUDA_CHECK(hipMalloc((void **) &align_pos_d, deformModel.dim * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **) &result_pos_d, deformModel.dim * sizeof(float)));
    // CuUDA Kernels run synchronously by default, to run asynchronously must explicitly specify streams

    // Calculate aligned positions
    calculateAlignedPositions(result_pos_d, align_pos_d, position_d, params, deformModel, scanPointCloud, cnpHandle);

    /*
     * Compute Point Pairs (Correspondances)
     */
    PointPair point_pair{
            .mesh_position_d=result_pos_d,
            .mesh_positoin_before_transform_d=align_pos_d,
            .ref_position_d=scanPointCloud.scanLandmark_d,
            .mesh_corr_inds_d=nullptr,
            .ref_corr_inds_d=nullptr,
            .point_count=scanPointCloud.numLmks
    };
    CUDA_MALLOC(&point_pair.mesh_corr_inds_d, static_cast<size_t>(scanPointCloud.numLmks));
    CUDA_MALLOC(&point_pair.ref_corr_inds_d, static_cast<size_t>(scanPointCloud.numLmks));

    calculateLandmarkIndices<<<1,scanPointCloud.numLmks>>>
                                 (point_pair.mesh_corr_inds_d, point_pair.ref_corr_inds_d, deformModel, scanPointCloud);

    // Calculate residual & jacobian for Landmarks
    calculatePointPairLoss(residual, fa1Jacobian, fa2Jacobian, ftJacobian, fuJacobian, point_pair,
                           params, deformModel, c_residuals, c_jacobians, weight, isJacobianRequired);

    CUDA_CHECK(hipFree(align_pos_d));
    CUDA_CHECK(hipFree(result_pos_d));
    CUDA_FREE(point_pair.mesh_corr_inds_d);
    CUDA_FREE(point_pair.ref_corr_inds_d);
}

void calculateGeometricLoss(float *residual, float *fa1Jacobian, float *fa2Jacobian, float *ftJacobian, float *fuJacobian,
                            float *position_d, hipblasHandle_t cnpHandle, const C_Params params,
                            const C_PcaDeformModel deformModel, const C_ScanPointCloud scanPointCloud,
                            C_Residuals c_residuals, C_Jacobians c_jacobians, const float searchRadius, const float weight,
                            const bool isJacobianRequired) {
    float *align_pos_d, *result_pos_d;
    const int num_residuals = c_residuals.numResuduals;

    // Allocate memory for Rigid aligned positions
    CUDA_CHECK(hipMalloc((void **) &align_pos_d, deformModel.dim * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **) &result_pos_d, deformModel.dim * sizeof(float)));
    // CuUDA Kernels run synchronously by default, to run asynchronously must explicitly specify streams

    /*
     * Compute Loss
     */
    // Calculate aligned positions
    calculateAlignedPositions(result_pos_d, align_pos_d, position_d, params, deformModel, scanPointCloud, cnpHandle);

    /*
     * Compute Point Pairs (Correspondances)
     */
    PointPair point_pair{
            .mesh_position_d=result_pos_d,
            .mesh_positoin_before_transform_d=align_pos_d,
            .ref_position_d=scanPointCloud.scanPoints_d,
            .mesh_corr_inds_d=nullptr,
            .ref_corr_inds_d=nullptr,
            .point_count=0
    };

    float* distance_d;
    int* numCorr_d; // TODO: Move to find_mesh_to_scan_corr, reference point_pair.point_count instead
    float radius = searchRadius;

    CUDA_MALLOC(&point_pair.mesh_corr_inds_d, static_cast<size_t>(num_residuals));
    CUDA_MALLOC(&point_pair.ref_corr_inds_d, static_cast<size_t>(num_residuals));
    CUDA_MALLOC(&distance_d, static_cast<size_t>(num_residuals));
    CUDA_MALLOC(&numCorr_d, static_cast<size_t>(1));

    find_mesh_to_scan_corr(point_pair.mesh_corr_inds_d, point_pair.ref_corr_inds_d, distance_d, numCorr_d,
                           result_pos_d, deformModel.dim, scanPointCloud, radius, num_residuals);

    CUDA_CHECK(hipMemcpy(&point_pair.point_count, numCorr_d, sizeof(int), hipMemcpyDeviceToHost));
    // TODO: Move to find_mesh_to_scan_corr and int* numCorr_d
    if (point_pair.point_count > num_residuals/3){
        point_pair.point_count = num_residuals/3;
    }

    /*******************
     * Calculate residual & jacobian for PointPairs
     *******************/
    calculatePointPairLoss(residual, fa1Jacobian, fa2Jacobian, ftJacobian, fuJacobian, point_pair,
                           params, deformModel, c_residuals, c_jacobians, weight, isJacobianRequired);

    CUDA_CHECK(hipFree(align_pos_d));
    CUDA_CHECK(hipFree(result_pos_d));
    CUDA_CHECK(hipFree(distance_d));
    CUDA_CHECK(hipFree(numCorr_d));
    CUDA_FREE(point_pair.mesh_corr_inds_d);
    CUDA_FREE(point_pair.ref_corr_inds_d);
}
